#include "hip/hip_runtime.h"
#define DEBUG

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "LTS.h"

#include <chrono> 
#include <iostream>
#include <fstream>

//Step 0: Set the correct (un)stable blocks, all the blocks the blocks, that
// have atleast one state are in the beginning unstable.
__global__ void set_stable(int N, bool* stable, int* block){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        stable[block[i]] = false;
    }
}


// Step 1: reset mark and pick a block
__global__ void pick_block(int N, bool* stable, bool* mark, int* current_block) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Reset the markings of previous round.
    if(i < N) {
        mark[i] = false;

        if(!stable[i])
            // atomicCAS(current_block, -1, i);
            *current_block = i;
   }
}
//Step 1a: reset marks

// Step 2: Mark the states which can reach the current block 
// && set the current block to stable
__global__ void mark(int M, int* source, int* target, int* order, int* marks_offset,
    bool* stable, bool* marks, int* current_block, int* block) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < M) {
        if(block[target[i]] == *current_block) {
            // Represents the transition source[i] ->labels[i] target[i]
            marks[marks_offset[source[i]] + order[i]] = true;
       }
    }

    //Set current block to stable
    if (i < 1 && *current_block != -1)
        stable[*current_block] = true;
}

// Step 3: Check for every transition if markings between leader is different
// and elect it as a new leader
__global__ void compare_markings(int M, int* source, int* order, int* marks_offset,
    bool* mark, bool* marks, int* block, int* next_number){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < M){
        if( marks[marks_offset[      source[i] ] + order[i]] != 
            marks[marks_offset[block[source[i]]] + order[i]]){

            mark[source[i]] = true;
            next_number[block[source[i]]] = source[i];
        }
    }
}

// Step 4: Split the block, update the block of the split off states
__global__ void split(int N, bool* stable,
            bool* mark, int* block, int* next_number, int* current_block) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        if(mark[i]) {
            stable[block[i]] = false;
            block[i] = next_number[block[i]];
            stable[block[i]] = false;
            stable[*current_block] = false;
       }
   }
}


int run_bisum_lab(int N, int M, int L, int* source, int* target, int* order, 
    int* block, int* marks_offset, int marks_length){

    //Setting the block sizes (threads per block) and nr of block
    int threads_N = 32;
    int blocks_N = (N + threads_N -1) / threads_N;

    int threads_M = 32;
    int blocks_M = (M + threads_M -1) / threads_M;
    
    //All states have a mark and marks array, also it has the block number
    bool *mark_d;
    bool *marks_d;
    gpuErrchk( hipMalloc((void**)&mark_d, sizeof(bool) * N) );
    gpuErrchk( hipMalloc((void**)&marks_d, sizeof(bool) * marks_length) );

    // All block have a next_number (which is the next leader)
    // and indicate if they are stable
    int* next_number_d;
    bool* stable_d;
    gpuErrchk( hipMalloc((void**)&next_number_d, sizeof(int) * N) );
    gpuErrchk( hipMalloc((void**)&stable_d, sizeof(bool) * N) );

    gpuErrchk( hipMemset(stable_d, 1, sizeof(bool) * N) );
    set_stable<<<blocks_N, threads_N>>>(N, stable_d, block);

    //The current block, undefined (-1) in the beginning
    int c = -1;
    int *c_d;
    gpuErrchk( hipMalloc((void**)&c_d, sizeof(int)) );
    gpuErrchk( hipMemcpy(c_d, &c, sizeof(int), hipMemcpyHostToDevice) );

    chrono::time_point<chrono::high_resolution_clock> start, end;

    start = chrono::high_resolution_clock::now();

    int iter = 0;
    // Executing kernel
    do {
        iter++;
        //Set current block to undefined
        c = -1;
        gpuErrchk( hipMemcpy(c_d, &c, sizeof(int), hipMemcpyHostToDevice) );

        // Step1: Pick the block to split
        pick_block<<<blocks_N, threads_N>>>(N, stable_d, mark_d, c_d);
        //Step 1a: reset marks
        gpuErrchk( hipMemset(marks_d, 0, sizeof(bool) * marks_length) );

        //Loop over the transitions to mark with the current block.
        mark<<<blocks_M, threads_M>>>(M, source, target, order, marks_offset, 
            stable_d, marks_d, c_d, block);

        //Compare markings and elect new leaders
        compare_markings<<<blocks_M, threads_M>>>(M, source, order, marks_offset, 
            mark_d, marks_d, block, next_number_d);

        //Split of the marked block
        split<<<blocks_N, threads_N>>>(N, stable_d, mark_d, block, next_number_d, c_d);

        //Get back the current block
        gpuErrchk( hipMemcpy(&c, c_d, sizeof(int), hipMemcpyDeviceToHost) );

        if(iter == 1)
        {
            end = chrono::high_resolution_clock::now();
            auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
            double time = duration.count() / 1000.0;
            printf("iter: %i, c: %i time: %g\n", iter, c, time);
        }
#ifdef DEBUG2        
        printf("iter: %i, c: %i\n", iter, c);
#endif
    } while( c != -1 && iter < 10*N );
  
    if(c != -1){
        cout << "WARNING: We passed a reasonable number of iterations("<< iter <<"), but we are not stable yet." << endl;
        return -1;
    }


    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Deallocate device memory
    gpuErrchk( hipFree(marks_d) );
    gpuErrchk( hipFree(mark_d) );
    gpuErrchk( hipFree(next_number_d) );
    gpuErrchk( hipFree(stable_d) );
    gpuErrchk( hipFree(c_d) );

    return iter;
}

int main(int argc, char *argv[]){
    LTS data;
    bool check = false;
    bool time = false;
    bool out = false;
    string out_fn;
    string in_fn;
    chrono::time_point<chrono::high_resolution_clock> start, end;
    double time_load, time_preprocess, time_alg, time_total;
    int blocks_remaining;
    
    string s_check ("--check");
    string s_time ("--time");
    string s_out ("--out");
    if(argc > 2){
        for(int i =2; i < argc; i++){
            if(s_check.compare(argv[i]) == 0)
                check = true;
            if(s_time.compare(argv[i]) == 0)
                time = true;
            if(s_out.compare(argv[i]) == 0){
                out = true;
                time = true;
                i++;
                if(i < argc)
                    out_fn = argv[i];
                else{
                    printf("Need output file\n");
                    exit(1);
                }
            }
        }
    }
    if(time)
        start = chrono::high_resolution_clock::now();



    if(argc < 1){
        exit(1);
        printf("Need input file\n");
    } else{
        in_fn = argv[1];
        data.Init(in_fn);
    }

    if(time){
        end = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
        time_load = duration.count() / 1000.0;
        start = chrono::high_resolution_clock::now();
        printf("Input done in %g\n", time_load);
    }

    data.init_device();
    data.preprocess();

    if(time){
        end = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
        time_preprocess = duration.count() / 1000.0;
        start = chrono::high_resolution_clock::now();
        printf("Preprocess done in %g\n", time_preprocess);
    }

    if(check){
        data.print_states(10);
        data.print_transitions(10);
    }

    int iter = run_bisum_lab(data.N, data.M, data.L, data.source_d,
        data.target_d, data.order_d, data.block_d, data.marks_offset_d,
        data.marks_length);

    if(time){
        end = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
        time_alg = duration.count() / 1000.0;
        start = chrono::high_resolution_clock::now();
        printf("Alg done in %g\n", time_alg);
    }

    if(time){
        time_total = time_load + time_preprocess + time_alg;
        printf("------------Timings (ms)------------\n");
        printf("Iter | Loading   | Preprocess | Algorithm | Total \n");
        printf("%d, %g, %g, %g, %g\n",
         iter, time_load, time_preprocess, time_alg, time_total);
    }

    if(check || out){
        vector<int> block(data.N);

        gpuErrchk( hipMemcpy(block.data(), data.block_d, sizeof(int) * data.N, hipMemcpyDeviceToHost) );

        sort(block.begin(), block.end());
        vector<int> unique_count;
        unique_count.clear();
        unique_copy(block.begin(), block.end(), back_inserter(unique_count));
        blocks_remaining = unique_count.size();

        printf("Remaing blocks: %d\n", blocks_remaining);
    }

    

    if(out){
        ofstream myfile;
        myfile.open (out_fn, ios_base::app);
        myfile << in_fn << "," << iter << "," <<
         time_load << "," << time_preprocess << "," <<
         time_alg << "," << time_total << "," <<
         blocks_remaining <<
         endl;
        myfile.close();
    }


    return 0;
}